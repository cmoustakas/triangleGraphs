#include "hip/hip_runtime.h"

/**
   
    Author : Chares Moustakas 
    AEM :    8860 
    E-mail:  cmoustakas@ece.auth.gr
    Course : Parallel & Distributed Systems
    Profs:   
	     Nikolaos Pitsianis , pitsiani@ece.auth.gr
	     Dimitrios Floros   , fcdimitri@auth.gr

**/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

float *readAdjacency(int N,char*filename);
int minimum(int a,int b);


hipsparseOperation_t trans = HIPSPARSE_OPERATION_NON_TRANSPOSE ;
hipsparseDirection_t direction = HIPSPARSE_DIRECTION_ROW ;


	

__global__ void hadamardKernel(int*csrRowPtrA,int*csrColIdxA,int* csrRowPtrC,int*csrColIdxC,int*csrRowPtrV,int*csrColIdxV,int N,int rounds,int* lastPtr,int top){

    int offset = rounds*513;
    int uId = threadIdx.x + blockIdx.x*blockDim.x + offset ;
    int columnCnt; 
    int row;
    int tempArray[10];
    int host;
    int limit = top + rounds;
    



    if(uId == 0 ){csrRowPtrV[uId] = 0;return;}
    else if(uId < N+1){
   
	
	for(int i =0;i<10;i++)tempArray[i]=0;	


	row = uId-1;	
	int lastPointA,startPointA;
	int lastPointC,startPointC;

	startPointA = csrRowPtrA[row]-1; 
	startPointC = csrRowPtrC[row]-1; 

	lastPointA = csrRowPtrA[uId]-2; 
	lastPointC = csrRowPtrC[uId]-2; 

	int elementA = csrColIdxA[startPointA];  
	int elementC = csrColIdxC[startPointC];  
	
	if(uId == offset && rounds>0)csrRowPtrV[uId-1] = lastPtr[0];
        csrRowPtrV[uId] = 0;
        columnCnt = 0;

	while(1){		
		if(elementA > csrColIdxC[lastPointC] || elementC > csrColIdxA[lastPointA] || startPointA > lastPointA || startPointC > lastPointC)break;
		else if(elementA>elementC)startPointC++;
		else if(elementA<elementC)startPointA++;
		else if(elementA==elementC){
			startPointC++;
			startPointA++;		
			tempArray[columnCnt] = elementA-1;
			columnCnt++;	
			csrRowPtrV[uId]++;
		}
		elementA = csrColIdxA[startPointA];
		elementC = csrColIdxC[startPointC];			
			
	}
     
   
     }
     host = csrRowPtrV[uId];
     
     __syncthreads();
     if(uId==limit){

	if(uId > N)limit = N;
        
		
	for(int i=offset; i < limit+1; i++)csrRowPtrV[i] += csrRowPtrV[i-1];
	lastPtr[0] = csrRowPtrV[limit]; 		//[-][-] Caution suspect overflow storage
	 
	}
     
     __syncthreads(); 
     if(uId<N+1 && uId > 0){
	for(int i=0;i<host;i++)
             csrColIdxV[csrRowPtrV[uId-1] + i] = tempArray[i];
     	}


} 






int main(int argc,char*argv[]){


   if(argc !=3){ printf("Usage : ./triangs -arg[1] = index -arg[1] = filename.txt \n");return 1;}

    // --- Host side Adjacency dense matrix
   int ind = atoi(argv[1]);
   int N = pow(2,ind);
   //printf("Init Matrix \n {Rows x Columns} = %d x %d \n",N,N);
   char* filename = argv[2];
   float *h_A_dense = (float*)malloc(N * N * sizeof(float));
   h_A_dense = readAdjacency(N,filename);
  
    int rounds = N/513 + 1;
    	


//  Initialize cuSPARSE
    hipsparseHandle_t handle;    

    
// SetUp My Descriptor
    hipsparseMatDescr_t descr ;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

// SetUp  Attributes :::
    hipError_t cudaStat = hipSuccess ;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS ;
    status = hipsparseCreate(&handle);
    assert(status == HIPSPARSE_STATUS_SUCCESS);

//Generate Kernel Streams [+]
       
    hipStream_t *kernel_stream;
    kernel_stream = (hipStream_t*)malloc(rounds*sizeof(hipStream_t));

    for(int i = 0;i<rounds;i++){
        cudaStat = hipStreamCreate(&kernel_stream[i]);
    	assert(cudaStat == hipSuccess);
    }


// Generate MemCopy Streams [+]

     hipStream_t cpyStream[6];
     for(int i = 0;i<6;i++){
	cudaStat = hipStreamCreate(&cpyStream[i]);
	assert(cudaStat == hipSuccess);
     }



// Setup My Device Matrix
    float *device_dense ;
    hipMalloc(&device_dense,N*N*sizeof(float));
    hipMemcpyAsync(device_dense,h_A_dense,N*N*sizeof(float),hipMemcpyHostToDevice,cpyStream[0]);
    free(h_A_dense);

    int nnz = 0;
    const int lda = N  ;

//Setup Non Zero Elements Per Row OF Matrix - Device Side
    int *device_nnz ;
    hipMalloc(&device_nnz,N*sizeof(int));
    status  =  hipsparseSnnz(handle,direction,N,N,descr,device_dense,lda,device_nnz,&nnz);
    assert(status == HIPSPARSE_STATUS_SUCCESS);
	
// Init SPARSE Matrix


    int* deviceRowptr;
    int* deviceColVector;
    float* deviceValVector;

    hipMalloc(&deviceRowptr,(N+1)*sizeof(int));
    hipMalloc(&deviceColVector,nnz*sizeof(int));
    hipMalloc(&deviceValVector,nnz*sizeof(float));

    hipsparseSdense2csr(handle,N,N,descr,device_dense,lda,device_nnz,deviceValVector,deviceRowptr,deviceColVector);
    hipFree(device_dense);



   /**
        Calculation of C =  A * A in csr format 
	[+][+][+][+][+][+][+][+][+][+][+]
   **/
   


   int baseC,nnzC ;
   int *nnzTotalDevHostPtr = (int*)malloc(sizeof(int));
   hipsparseSetPointerMode(handle,HIPSPARSE_POINTER_MODE_HOST);
   
   int* csrRowPtrC ;
   hipMalloc(&csrRowPtrC,sizeof(int)*(N+1));

   hipsparseMatDescr_t descrC ; 									// SetUp descriptor for the C array
   hipsparseCreateMatDescr(&descrC);
   hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ONE);
   
   
   hipsparseXcsrgemmNnz(handle,trans,trans,N,N,1, 						// Handling And attributes ..
		       	   	   descr,nnz,deviceRowptr,deviceColVector, 			// Attributes Of Array A 
		       	   	   descr,nnz,deviceRowptr,deviceColVector,descrC, 		// Attributes Of Array B (but yall know B = A :O ) and the descriptor for C
		       	   	   csrRowPtrC,nnzTotalDevHostPtr);
                       										// Output of the function ** Total NonZero Values And Row Pointer Matrix 

    if(NULL != nnzTotalDevHostPtr)
	nnzC = *nnzTotalDevHostPtr ;
    else{
    	hipMemcpyAsync(&nnz,csrRowPtrC+N,sizeof(int),hipMemcpyDeviceToHost,cpyStream[1]);
    	hipMemcpyAsync(&baseC,deviceRowptr,sizeof(int),hipMemcpyDeviceToHost,cpyStream[2]);
    	nnz = nnz -baseC ;
        									// So now you got the host of non zero elements in your new array .. Lets calculate it then !
    }
    int* csrColIdxC ;
    float* csrValueC ;	
    
    hipMalloc(&csrColIdxC,sizeof(int)*nnzC);
    hipMalloc(&csrValueC,sizeof(float)*nnzC);
    
    clock_t start = clock();    

    hipsparseScsrgemm(handle,trans,trans,N,N,1,
	                descr,nnz,deviceValVector,deviceRowptr,deviceColVector,
	                descr,nnz,deviceValVector,deviceRowptr,deviceColVector,
	                descrC,csrValueC,csrRowPtrC,csrColIdxC);
   

   
   if(nnzC == 0){
	printf("[-][-][-][-] Number oF Triangles is Zero \n\n");
	clock_t stop = clock();
	float time = (float)(stop-start)/CLOCKS_PER_SEC;
		
	printf("Execution time : %f sec \n",time); 
	return 0;	
    
    }   	

//	Print The Csr In Dense ..


   float* C_Array = (float*)malloc(N*N*sizeof(float));
   float* device_C_Array ;
   hipMalloc(&device_C_Array,N*N*sizeof(float));
   hipsparseScsr2dense(handle,N,N,descrC,csrValueC,csrRowPtrC,csrColIdxC,device_C_Array,N);
   hipMemcpyAsync(C_Array,device_C_Array,sizeof(float)*N*N,hipMemcpyDeviceToHost,cpyStream[3]); 
   hipFree(device_C_Array);
   

   
/**

deviceRowptr     -----------+
deviceColVector             |-----> Vector A  
deviceValVector  -----------+


csrRowPtrC       -----------+
csrColIdxC                  |-----> Vector C = A x A
csrValueC        -----------+

hadamardKernel(float*csrValueA,int*csrRowPtrA,int*csrColIdxA,float* csrValueC,float* csrValueV,int* csrRowPtrC,int* csrRowPtrV,int*csrColIdxC,int* csrColIndxV)

   [+][+][+][+][+][+][+]  So Now I Need To Multiply (Hadamard) Elementwisely C And A. 
   

**/
    
    
    int *d_csrRowPtrV,*d_csrColIndxV,*d_lastPtr;
    
    hipMalloc(&d_lastPtr,sizeof(int));
    hipMalloc(&d_csrRowPtrV,(N+1)*sizeof(int));
    int minNnz = minimum(nnzC,nnz);
    hipMalloc(&d_csrColIndxV,minNnz*sizeof(int));

    
   
    int top;
    if(N>513) top = 512;
    else top = N;
    
    int blocks = 1;
    int threads = top+1;


    
    for(int i = 0;i<rounds;i++){  
        hadamardKernel<<<blocks,threads,0,kernel_stream[i]>>>(deviceRowptr,deviceColVector,
					              csrRowPtrC,csrColIdxC,
						      d_csrRowPtrV,d_csrColIndxV,
						      N,i,d_lastPtr,top);
        
	top += 512 ;
        hipStreamSynchronize(kernel_stream[i]);
    }
  
   
  
    for(int i =0;i<rounds;i++)cudaStat = hipStreamDestroy(kernel_stream[i]);
    hipError_t err = hipGetLastError();
    if(cudaStat != hipSuccess)printf("[-][-] Error : %s \n",hipGetErrorString(err));
    

    int*h_csrRowPtrV = (int*)malloc((N+1)*sizeof(int));
    int *h_csrColIdxV = (int*)malloc(minNnz*sizeof(int));
    hipMemcpyAsync(h_csrColIdxV,d_csrColIndxV,minNnz*sizeof(int),hipMemcpyDeviceToHost,cpyStream[4]);
    hipMemcpyAsync(h_csrRowPtrV,d_csrRowPtrV,(N+1)*sizeof(int),hipMemcpyDeviceToHost,cpyStream[5]);
    
    //hipStreamSynchronize(cpyStream[5]);
    


//Pretty Interesting Line, Uncomment If you are Enemy Of Performance


//  hipDeviceReset();





    
  /**
	[+] Number Of Triangles Calculation
  **/
   
   long unsigned int host_cnt,host,col;
   long long unsigned int  row ;
    long long int sum=0;
    
    for(long int i = 0;i<N;i++){

	 host_cnt = 0;	
	 host = h_csrRowPtrV[i+1]-h_csrRowPtrV[i];
	 while(host_cnt<host){
	       row =N*h_csrColIdxV[h_csrRowPtrV[i]+host_cnt];
	       col=i;
	       sum += C_Array[row + col];
	       host_cnt++;
	
	}
     }
	     
	      
    long int numOfT = (int)(sum/6);
  


    clock_t stop = clock();
    float time = (float)(stop-start)/CLOCKS_PER_SEC;	
     
    hipDeviceReset();

    printf("\n[+][+][+][+][+] Number Of Triangles : %ld \n",numOfT);
    printf("Execution time : %f sec \n",time);


    return 0;

}



int minimum(int a,int b){
   if(a<b)return a;
   else  return b;
}



float *readAdjacency(int N,char* filename){
    FILE *fp;
    float *array,value;
    array = (float*)malloc(N*N*sizeof(float));
    int counter=0;
    fp = fopen(filename,"r");
    if(fp==NULL){return NULL;}
    int i=10,offset = 0;

    while(counter<N*N){
	if(i == EOF){printf("Not big enough txt file");exit(0);}        
	while(offset<N){	
    		i=fscanf(fp,"%f",&value);
    		array[counter+offset] = value;
    	        offset++;
        }
	i = fscanf(fp,"%*[^\n]\n");	
	counter = counter+N;
	offset = 0;
    }
    
    
    if(N<17){
        printf("Array = [");
        for(int k = 0 ;k<counter ;k++){
	    if(!(k%N))printf("\n");        
	    printf("%f ",array[k]);
	
        } 
        printf("] \n");
    }
    
    fclose(fp);
    return array;   
}

